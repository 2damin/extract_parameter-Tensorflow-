#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include <sstream>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "changeDatatype.cuh"

using namespace std;

__global__ void changeType(float* srcData, float* dstData, int n, int c, int h, int w, int filtersPerThread_x, int filtersPerThread_y) {
	const int idxCol = threadIdx.y + blockDim.y*blockIdx.y;
	const int idxRow = threadIdx.x + blockDim.x*blockIdx.x;
	int maxBlock = (n * c) / (filtersPerThread_x * filtersPerThread_y);
	int idxBlock = (int)fminf((float)(blockIdx.y * gridDim.x + blockIdx.x), (float)(maxBlock));

	const int idxfilterW = threadIdx.x % w;
	const int idxfilterH = threadIdx.y % h;
	int threadChannelX = threadIdx.x / w;
	int threadChannelY = threadIdx.y / h;
	int idxChannel_a =idxBlock * filtersPerThread_x * filtersPerThread_y + threadChannelY *filtersPerThread_x + threadChannelX;
	int idxChannel = idxChannel_a % c;
	int idxN = (int)fminf((float)(idxChannel_a / c), (float)(n-1));	

	dstData[idxN * (c * w* h) + idxChannel * (w*h) + idxfilterH * w + idxfilterW] = srcData[idxfilterH * (n * c * w) + idxfilterW * (c * n) + idxChannel * n + idxN];

}


void changeDataType(float* srcData, float* dstData, int n, int c, int h, int w) {
	
	int filtersPerThread_x = 30 / w;
	int filtersPerThread_y = 30 / h;

	int totalBlocks = (c * n) / (filtersPerThread_x * filtersPerThread_y) + 1;
	int numBlock_y = totalBlocks / 255 + 1;

	dim3 numOfBlocks(255, numBlock_y, 1);
	dim3 threadsPerBlock(30, 30, 1);
	changeType <<< numOfBlocks, threadsPerBlock >> > (srcData, dstData, n, c, h, w, filtersPerThread_x, filtersPerThread_y);
}